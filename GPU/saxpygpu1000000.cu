#include "hip/hip_runtime.h"
#include <stdlib.h>
#include <stdio.h>
#include <string.h>
#include <time.h>

#include <cutil.h>
#include <hip/hip_runtime.h>

#define A 3

/***    Initialisation des vecteurs    ***/
void initab(float * tab, float val, int SIZE) {
   int k;
   for (k=0; k<SIZE; k++)   {
   tab[k]= val;
   }
}

__global__ void saxpy_gpu(float *tabX, float * tabY, int sizemax) {
   // Parametrer l'operation avec threadIdx.x;
    int k = threadIdx.x + blockDim.x * (blockIdx.x + blockIdx.y * gridDim.x);

    if (k<sizemax) {
    tabY[k] = tabX[k]*A + tabY[k];
    }
}

/***    Affichage des vecteurs    ***/
void affitab(char * chaine,float * tab, int SIZE)
{
   int k;
   printf("\nLes 16 premiers de %s: \n",chaine);
   for (k=0; k<16; k++) printf("%.2f ",tab[k]);
   printf("\nLes 16 derniers: \n");
   for (k=SIZE-16; k<SIZE; k++) printf("%.2f ",tab[k]);
   printf("\n");
}

////////////////////////////////////////////////////////////////////////////////
// Program main
////////////////////////////////////////////////////////////////////////////////
int
main( int argc, char** argv) 
{ 

    float * tablox_d, * tablox_h;
    float * tabloy_d, * tabloy_h;

    int nbelements=35000000;//valeur initiale
    int maxdim = 65535;
    int sizeblock = 512;
    
    int nblock = (nbelements/sizeblock)+1;
    int nbligne = (nblock/maxdim)+1;
      /** Initialisation de  nbthreadbyblock et nbblockbygrid**/

    dim3 nbthreadbyblock(sizeblock);
    
    dim3 nbblockbygrid(maxdim,nbligne);

    printf("Debut du programme avec %d nbelements \n",nbelements);

      /** Allocation memoire sur le host(CPU) **/
    tablox_h=(float *) malloc(sizeof(float) * nbelements);
    initab(tablox_h, 1., nbelements);
    tabloy_h=(float *) malloc(sizeof(float) * nbelements);
    initab(tabloy_h, 2., nbelements);


      /** Affichage initial **/
    printf("Affichage initial\n");
    affitab("tabloy_h",tabloy_h, nbelements);

      /** Allocation memoire sur le device(GPU) **/
    // Utilisation de hipMalloc
    CUDA_SAFE_CALL(hipMalloc((void**) &tablox_d, sizeof(float) * nbelements));
    CUDA_SAFE_CALL(hipMalloc((void**) &tabloy_d, sizeof(float) * nbelements));

      /** Transfert memoire du host vers le device **/
    CUDA_SAFE_CALL(hipMemcpy((void*)tablox_d, (void*) tablox_h, sizeof(float) * nbelements, hipMemcpyHostToDevice));
    CUDA_SAFE_CALL(hipMemcpy((void*)tabloy_d, (void*) tabloy_h, sizeof(float) * nbelements, hipMemcpyHostToDevice));

      /** Lancement du kernel **/
    saxpy_gpu <<<nbblockbygrid, nbthreadbyblock>>> (tablox_d, tabloy_d, nbelements);

      /** Synchronisation **/
    CUDA_SAFE_CALL(hipDeviceSynchronize());

      /** Transfert memoire du device vers le host **/
    //CUDA_SAFE_CALL(hipMemcpy((void*)tablox_h, (void*) tablox_d, sizeof(float) * nbelements, hipMemcpyDeviceToHost));
    CUDA_SAFE_CALL(hipMemcpy((void*)tabloy_h, (void*) tabloy_d, sizeof(float) * nbelements, hipMemcpyDeviceToHost));

      /** Affichage du resultat **/
    printf("Affichage du resultat\n");
    affitab("tabloy_h",tabloy_h, nbelements);

      /** Liberation memoire **/
        hipFree(tablox_d); hipFree(tabloy_d);
        free(tablox_h); free(tabloy_h);  

    printf("Fin du programme\n");
    return EXIT_SUCCESS;

}


