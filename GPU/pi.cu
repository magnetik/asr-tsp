#include "hip/hip_runtime.h"
#include <stdlib.h>
#include <stdio.h>
#include <string.h>
#include <time.h>

#include <cutil.h>
#include <hip/hip_runtime.h>

#define A 3

__global__ void saxpy_gpu(float *tab, int nbintervals) {
   // Parametrer l'operation avec threadIdx.x;
    int k = threadIdx.x + blockDim.x * (blockIdx.x + blockIdx.y * gridDim.x);
    float delta = (float) 1.0/nbintervals;
    if (k<nbintervals) {
        tab[k] = (4.0/(1.0+(k*delta)*(k*delta))) * delta;
    }
}

/***    Affichage des vecteurs    ***/
void affitab(char * chaine,float * tab, int SIZE)
{
   int k;
   printf("\nLes 16 premiers de %s: \n",chaine);
   for (k=0; k<16; k++) printf("%.2f ",tab[k]);
   printf("\nLes 16 derniers: \n");
   for (k=SIZE-16; k<SIZE; k++) printf("%.2f ",tab[k]);
   printf("\n");
}

////////////////////////////////////////////////////////////////////////////////
// Program main
////////////////////////////////////////////////////////////////////////////////
int
main( int argc, char** argv) 
{ 

    float * tablo_d, * tablo_h;

    int nbintervals=1000;//valeur initiale
    int maxdim = 65535;
    int sizeblock = 512;
    
    float delta = (float) 1.0/nbintervals;
    
    int nblock = (nbintervals/sizeblock)+1;
    int nbligne = (nblock/maxdim)+1;
    
    int i;
    float pi = 0.;
      /** Initialisation de  nbthreadbyblock et nbblockbygrid**/

    dim3 nbthreadbyblock(sizeblock);
    
    dim3 nbblockbygrid(maxdim,nbligne);

      /** Allocation memoire sur le host(CPU) **/
    tablo_h=(float *) malloc(sizeof(float) * nbintervals);

      /** Allocation memoire sur le device(GPU) **/
    CUDA_SAFE_CALL(hipMalloc((void**) &tablo_d, sizeof(float) * nbintervals));

      /** Lancement du kernel **/
    saxpy_gpu <<<nbblockbygrid, nbthreadbyblock>>> (tablo_d, nbintervals);

      /** Synchronisation **/
    CUDA_SAFE_CALL(hipDeviceSynchronize());

      /** Transfert memoire du device vers le host **/
    CUDA_SAFE_CALL(hipMemcpy((void*)tablo_h, (void*) tablo_d, sizeof(float) * nbintervals, hipMemcpyDeviceToHost));


    //Réduction cpu
    for (i=0; i<nbintervals; i++) {
        pi += tablo_h[i];
    }
    pi = pi - ( (4. + 2.) * delta / 2.);
      /** Affichage du resultat **/
    printf("Affichage du resultat ; pi= %f\n",pi);
    

      /** Liberation memoire **/
        hipFree(tablo_d); 
        free(tablo_h);   

    printf("Fin du programme\n");
    return EXIT_SUCCESS;

}


