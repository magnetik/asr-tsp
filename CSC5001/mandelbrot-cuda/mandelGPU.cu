#include "hip/hip_runtime.h"
/*
 * CUDA Mandelbrot program 
 * 
 * This program computes and displays all or part of the Mandelbrot 
 * set.  By default, it examines all points in the complex plane
 * that have both real and imaginary parts between -2 and 2.  
 * Command-line parameters allow zooming in on a specific part of
 * this range.
 * 
 * Usage:
 *   mandelbrot maxiter 
 * where 
 *   maxiter denotes the maximum number of iterations at each point
 * 
 * Input:  none, except the optional command-line arguments
 * Output: a graphical display as described in Wilkinson & Allen,
 *   displayed using the X Window system, plus text output to
 *   standard output showing the above parameters.
 * 
 * 
 * Code originally obtained from Web site for Wilkinson and Allen's
 * text on parallel programming:
 * http://www.cs.uncc.edu/~abw/parallel/par_prog/
 * 
 * Reformatted and revised by B. Massingill and C. Parrot and C.Schuller
 */
#include <stdio.h>
#include <math.h>
#include <stdlib.h>
#include <unistd.h>
#include <X11/Xlib.h>
#include <X11/Xutil.h>
#include <X11/Xos.h>

#include <cutil.h>
#include <hip/hip_runtime.h>
//#ifdef WITH_DISPLAY
#include "mandelbrot-gui.h"     /* has setup(), interact() */
//#endif
/* Default values for things. */
#define N           2           /* size of problem space (x, y from -N to N) */
#define NPIXELS     600         /* size of display window in pixels */

/* Structure definition for complex numbers */
typedef struct {
  float r, i;
} complex ;

/* Shorthand for some commonly-used types */
typedef unsigned int uint;
typedef unsigned long ulong;

/* Description pixel */
typedef struct {
  uint col, row;
  ulong couleur; 
} Pixel ;


__global__ void mandelbrot_gpu(ulong *vect_d, ulong max_color, ulong min_color, float scale_r, float scale_i, float scale_color, int maxiter) {
    complex z, c;
    int col;
    int row;
    int k;
    float r_min = -N;
    float i_min = -N;
    float lengthsq, temp;
    ulong couleur;
        
   // Parametrer l'operation avec threadIdx.x;
    col = threadIdx.x + (blockIdx.x * blockDim.x);
    row = threadIdx.y + (blockIdx.y * blockDim.y);
    
      
      z.r = z.i = 0;
      
      /* Scale display coordinates to actual region  */
      c.r = r_min + ((float) col * scale_r);
      c.i = i_min + ((float) (NPIXELS-1-row) * scale_i);

      /* Calculate z0, z1, .... until divergence or maximum iterations */
      k = 0;
      do  {
	    temp = z.r*z.r - z.i*z.i + c.r;
	    z.i = 2*z.r*z.i + c.i;
	    z.r = temp;
	    lengthsq = z.r*z.r + z.i*z.i;
	    ++k;
      } while (lengthsq < (N*N) && k < maxiter);
      
      /* Scale color and display point */ 
      couleur = (ulong) ((k-1) * scale_color) + min_color;
      
      vect_d[col+row*NPIXELS]=couleur;
}
////////////////////////////////////////////////////////////////////////////////
// Main program
/////////////////////////////////////////////////////////////////////////////

int main(int argc, char *argv[]) {
  uint maxiter;
  float r_min = -N;
  float r_max = N;
  float i_min = -N;
  float i_max = N;
  uint width = NPIXELS;         /* dimensions of display window */
  uint height = NPIXELS;
  Display *display;
  Window win;
  GC gc;
  int setup_return;
  ulong min_color, max_color;
  float scale_r, scale_i, scale_color;
  uint k;
  ulong * vect_h;
  ulong * vect_d;
  unsigned int timer;
  int nbblockligne, nbblockcolonne;
  
  /* Check command-line arguments */
  if (argc < 2) {
    fprintf(stderr, "usage:  %s maxiter \n", argv[0]);
    return EXIT_FAILURE;
  }

  vect_h = (ulong *) malloc(sizeof(ulong) * NPIXELS * NPIXELS);
    /* Alocate memory on device */
  CUDA_SAFE_CALL(hipMalloc((void**) &vect_d, sizeof(ulong) * NPIXELS * NPIXELS));

  //printf("Debut du programme\n");
  /* Process command-line arguments */
  maxiter = atoi(argv[1]);
#ifdef WITH_DISPLAY
  /* Initialize for graphical display */
  setup_return = 
    setup(width, height, &display, &win, &gc, &min_color, &max_color);
  if (setup_return != EXIT_SUCCESS) {
    fprintf(stderr, "Unable to initialize display, continuing\n");
  }
#else
    min_color=0;
    max_color=16777215;
#endif
  /* Calculate and draw points */
  
  /* Compute factors to scale computational region to window */
  scale_r = (float) (r_max - r_min) / (float) width;
  scale_i = (float) (i_max - i_min) / (float) height;
  
  /* Compute factor for color scaling */
  scale_color = (float) (max_color - min_color) / (float) (maxiter - 1);
  
  /* Calcul */
  if ((NPIXELS % 200) ==0) 
    nbblockligne = NPIXELS/200;
  else 
    nbblockligne = (NPIXELS/200)+1;
  if ((NPIXELS % 2) ==0) 
    nbblockcolonne = NPIXELS/2;
  else 
    nbblockcolonne = (NPIXELS/2)+1;

  dim3 nbblockbygrid(nbblockligne, nbblockcolonne);
  dim3 nbthreadbyblock(200,2);
  cutCreateTimer(&timer); 
  cutStartTimer(timer);  
  
  //mandelbrot_gpu(ulong vect_d[][], ulong maxcolor, ulong mincolor, float scale_r, float scale_i, float scale_color, int maxiter)
  mandelbrot_gpu <<<nbblockbygrid, nbthreadbyblock>>> (vect_d, max_color, min_color, scale_r, scale_i, scale_color, maxiter);
  
  CUDA_SAFE_CALL(hipDeviceSynchronize());
  
    cutStopTimer(timer);
  printf("%f\n",cutGetTimerValue(timer));
  cutDeleteTimer(timer);
  
  CUDA_SAFE_CALL(hipMemcpy((void*)vect_h, (void*) vect_d, sizeof(ulong) * NPIXELS * NPIXELS, hipMemcpyDeviceToHost)); 

    //printf("Fin du calcul des pixels GPU\n");

#ifdef WITH_DISPLAY
     //printf("Debut affichage\n");
     for (k=0; k<(NPIXELS*NPIXELS); k++)

      if (setup_return == EXIT_SUCCESS) {
	XSetForeground (display, gc, vect_h[k]);
	XDrawPoint (display, win, gc, k%NPIXELS, k/NPIXELS);
      }
#endif
     //printf("Fin affichage\n");
     free(vect_h);
       hipFree(vect_d);
     //printf("Fin attente\n");

  return EXIT_SUCCESS;
}

