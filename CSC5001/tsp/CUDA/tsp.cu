#include "hip/hip_runtime.h"
/*
 * Sequential Mandelbrot program 
 * 
 * This program computes and displays all or part of the Mandelbrot 
 * set.  By default, it examines all points in the complex plane
 * that have both real and imaginary parts between -2 and 2.  
 * Command-line parameters allow zooming in on a specific part of
 * this range.
 * 
 * Usage:
 *   mandelbrot maxiter 
 * where 
 *   maxiter denotes the maximum number of iterations at each point
 * 
 * Input:  none, except the optional command-line arguments
 * Output: a graphical display as described in Wilkinson & Allen,
 *   displayed using the X Window system, plus text output to
 *   standard output showing the above parameters.
 * 
 * 
 * Code originally obtained from Web site for Wilkinson and Allen's
 * text on parallel programming:
 * http://www.cs.uncc.edu/~abw/parallel/par_prog/
 * 
 * Reformatted and revised by B. Massingill and C. Parrot and C.Schuller
 */
#include <stdio.h>
#include <math.h>
#include <stdlib.h>
#include <unistd.h>
#include <X11/Xlib.h>
#include <X11/Xutil.h>
#include <X11/Xos.h>

#include <cutil.h>
#include <hip/hip_runtime.h>

#include "tsp-types.h"

/* Shorthand for some commonly-used types */
typedef unsigned int uint;
typedef unsigned long ulong;

void genmap () 
{
    #define MAXX	100
    #define MAXY	100
    typedef struct
		    {
		     int x, y ;
		    } coor_t ;

    typedef coor_t coortab_t [MAXNBCITIES] ;
    coortab_t towns ;
    int i, j ;
    int dx, dy ;

     if (NbCities > MAXNBCITIES) {
      fprintf(stderr,"trop de villes, augmentez MAXNBCITIES dans tsp-types.h");
      exit(1);
     }

     srand (seed) ;

     for (i=0; i<NbCities; i++)
      {
       towns [i].x = rand () % MAXX ;
       towns [i].y = rand () % MAXY ;
      }

     for (i=0; i<NbCities; i++)
      {
       for (j=0; j<NbCities; j++)
        {
         /* Un peu r�aliste */
         dx = towns [i].x - towns [j].x ;
         dy = towns [i].y - towns [j].y ;
         distance [i][j] = (int) sqrt ((double) ((dx * dx) + (dy * dy))) ;
        }
      }
}


__global__ void tsp_gpu() {
   
}
////////////////////////////////////////////////////////////////////////////////
// Main program
/////////////////////////////////////////////////////////////////////////////

int main(int argc, char *argv[]) {
    uint32 ** Solution ;
    
    genmap(); 
    
    Solution = new uint32*[MAIN_NbIter] ;
    Solution[0] = new uint32[NbNode*MAIN_NbIter] ;

    return EXIT_SUCCESS;
}

